#include "hip/hip_runtime.h"
#include <hpc/sensors/Network.hpp>

#include <hip/hip_runtime.h>
#include <sys/time.h>

#include "network_kernel.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ void wait10kclocks() {
    clock_t start_clock = clock();

    while (clock() - start_clock < 10000);
}

__global__ void network_kernel(CyclicBuffer<int, 80> * bufs, hiprandState * state){
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState localState = state[id];

    int c = 0;
    bool random_bit = false;

    __syncthreads();
    while (++c < 20){
        random_bit = static_cast<bool>(hiprand(&localState) & 1);

        if (random_bit){
            bufs[id].push(static_cast<int>('A') + (c % ('Z' - 'A')));
        }

        __syncthreads();
    }

    state[id] = localState;
}

/*
__global__ void sensor_kernel() {
    value_type value;
    while (true) {
        get_messages();
        __synchronize();
        read_physical_data();
        __synchronize();
        send_messages();
    }
}*/
