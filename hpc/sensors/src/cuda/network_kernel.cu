#include "hip/hip_runtime.h"
#include <hpc/sensors/Network.hpp>

#include "network_kernel.cuh"

__global__ void network_kernel(int *data) {
    __shared__ bool running;

    running = true;
    // __syncthreads()
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    // unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    __syncthreads();
    data[x] = static_cast<int>('A') + x;
    __syncthreads();

    // while (running) {
    //     __syncthreads();

    // }
}

/*
__global__ void sensor_kernel() {
    value_type value;
    while (true) {
        get_messages();
        __synchronize();
        read_physical_data();
        __synchronize();
        send_messages();
    }
}*/
