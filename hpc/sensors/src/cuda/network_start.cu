#include "hip/hip_runtime.h"
#include <hpc/sensors/Network.hpp>

#include <unistd.h>

#include "network_kernel.cuh"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <cstdlib>
#include <ctime>

#include <iostream>
#include <iomanip>

__global__ void curand_setup_kernel(hiprandState * state, unsigned long seed = 0) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(seed, id, 0, &state[id]);
}

void network_start() {
    constexpr unsigned BLK_COUNT = 1;
    constexpr unsigned THREADS_PER_BLK = 16;

    thrust::device_vector<CyclicBuffer<int, CBUF_SIZE>> dev_v(BLK_COUNT * THREADS_PER_BLK);

    hiprandState * devStates;
    hipMalloc((void **)&devStates, BLK_COUNT * THREADS_PER_BLK * sizeof(hiprandState));

    srand(time(NULL));

    curand_setup_kernel<<<BLK_COUNT, THREADS_PER_BLK>>>(devStates, rand());
    network_kernel<<<BLK_COUNT, THREADS_PER_BLK>>>(thrust::raw_pointer_cast(dev_v.data()), devStates);

    thrust::host_vector<CyclicBuffer<int, CBUF_SIZE>> host_v = dev_v;

    std::cout << "wait 0.1 sec" << std::endl;
    usleep(100000);

    host_v = dev_v;

    for (auto & a : host_v) {
        std::cout << std::setw(3) << a.size() << ": ";

        while(!a.empty())
            std::cout << static_cast<char>(a.pop());

        std::cout << std::endl;
    }

    hipFree(devStates);
}
