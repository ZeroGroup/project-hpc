#include "hip/hip_runtime.h"
#include <hpc/sensors/Network.hpp>

#include <thrust/device_vector.h>

__global__ void kernel(int *data) {
    // __syncthreads()
   unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
   // unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

   data[x] = (static_cast<int>('A') + x);
}

/*
__global__ void sensor_kernel() {
    value_type value;
    while (true) {
        get_messages();
        __synchronize();
        read_physical_data();
        __synchronize();
        send_messages();
    }
}*/

#include <iostream>

void network_push(){

    thrust::device_vector<int> dev_v(16);
    // for (int i = 0; i < 16; ++i){
    //     int* raw_ptr;

    //     dev_v.push_back(raw_ptr);
    // }

    kernel<<<1,16>>>(thrust::raw_pointer_cast(dev_v.data()));

    for (int a : dev_v)
        std::cout << static_cast<char>(a) << std::endl;
}
